#include "hip/hip_runtime.h"
/*
/   Code by SM, based on customs.c from CS305
/   C++ version of the eventual CUDA implementation.
/   Simulates a day in the life of some customs agents, sequentially.
/   Change the constants to edit the parameters of the simulation.
*/
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include ""
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define MIN_CHILDREN 1 //default 1
#define MAX_CHILDREN 3 //default 3
#define CHILD_MOD1 4 //default 4
#define CHILD_MOD2 4 //default 4
#define CITIZEN_CHANCE 5 //default 5 (80%)
#define WORK_HOURS 8 //default 8
#define NORMAL_PAY 20 //default 20
#define OVERTIME_PAY 30 //default 30
#define NUM_AGENTS 20 //default 20
#define NUM_GROUPS 1000 //default 1000
#define SEED NULL //default NULL (0)

struct group {
    int adults;
    int children;
    bool usa;
    //thrust::device_vector<group*> group;
};

struct agent {
    int timecard;
    int avail;
    thrust::device_vector<group*> group;
};

struct stats {
    int total_time;
    int total_payroll;
    int avg_wait_time;
    int max_wait_time;
};

group* create_group() {
    group* g = new group;
    g->adults = MIN_CHILDREN + rand() % MAX_CHILDREN;
    g->children = (rand() % CHILD_MOD1) + (rand() % CHILD_MOD2) - 2;
    if (g->children < 0)
        g->children = 0;
    g->usa = ((rand() % CITIZEN_CHANCE) == 0) ? false : true;
    return g;
}

agent* create_agent() {
    agent* a = new agent;
    a->timecard = 0;
    a->avail = 0;
    return a;
}

/*__device__ void enqueue(agent* agt, group* grp) {
    if (agt == nullptr || grp == nullptr)
        return;

    agt->group.push_back(grp);
}

__device__ group* dequeue(agent* agt) {
    if (agt->group.front() == nullptr || agt == nullptr)
        return nullptr;
    group* grp = agt->group.back();
    agt->group.pop_back();
    return grp;
}*/

//this is parallel now
__global__ void calc_time(int total_time, thrust::device_vector<agent*> agents, agent* a, group* g) {
    int max_time = 0;
    int j = 0;

    int i = threadIdx.x;
    a = agents[i]; //broken???
    int agent_time = 0;
    g = a->group[0]; //broken???

    while (g != nullptr && j < NUM_AGENTS) {
        int temp = g->adults;
        if (!g->usa)
            temp *= 2;
        temp += (1 + g->children) / 2;
        agent_time += temp;
        delete(g);
        //g = dequeue(a);
        j++;
        g = a->group[j]; //broken???
    }

    if (agent_time > max_time)
        max_time = agent_time;
    if (max_time > total_time)
        total_time = max_time;
}

int calc_payroll(int time) {
    int cost = 0;

    if ((time / 60) <= WORK_HOURS)
        cost = (time / 60) * NORMAL_PAY * NUM_AGENTS;
    if ((time / 60) > WORK_HOURS) {
        cost = WORK_HOURS * NORMAL_PAY * NUM_AGENTS;
        cost += ((time / 60) - 8) * OVERTIME_PAY * NUM_AGENTS;
    }
    return cost;
}

__device__ int elapsed;

//let's do this!
int main() {
    srand(time(SEED));

    //
    thrust::host_vector<agent*> h_agents;
    thrust::fill(h_agents.begin(), h_agents.begin()+NUM_AGENTS, create_agent());
    for (int i = 0; i < NUM_AGENTS; i++) {
        thrust::fill((*h_agents[i]).group.begin(), (*h_agents[i]).group.begin()+NUM_GROUPS, create_group());
    }

    thrust::device_vector<agent*> d_agents = h_agents;

    group* h_g = (group*)malloc(sizeof(h_g));
    agent* h_a = (agent*)malloc(sizeof(h_a));
    int elapsed = 0;

    group* d_g;
    hipMalloc(&d_g, sizeof(d_g));
    hipMemcpy(d_g, h_g, sizeof(group*), hipMemcpyHostToDevice);

    agent* d_a;
    hipMalloc(&d_a, sizeof(d_a));
    hipMemcpy(d_a, h_a, sizeof(agent*), hipMemcpyHostToDevice);

    calc_time<<<1, NUM_AGENTS>>>(elapsed, d_agents, d_a, d_g);
    int payroll = calc_payroll(elapsed);
    int average = (elapsed / NUM_AGENTS);

    std::cout << "-- Simulation Parameters --\n" << std::endl;
    std::cout << "Minimum Number of Children: " << MIN_CHILDREN << "\n" << std::endl;
    std::cout << "Maximum Number of Children: " << MAX_CHILDREN << "\n" << std::endl;
    std::cout << "Child Mod 1: " << CHILD_MOD1 << "\n" << std::endl;
    std::cout << "Child Mod 2: " << CHILD_MOD2 << "\n" << std::endl;
    std::cout << "Citizen Chance Modifier: " << CITIZEN_CHANCE << "\n" << std::endl;
    std::cout << "Work Hours: " << WORK_HOURS << "\n" << std::endl;
    std::cout << "Normal Hourly Pay: " << NORMAL_PAY << "\n" << std::endl;
    std::cout << "Overtime Hourly Pay: " << OVERTIME_PAY << "\n" << std::endl;
    std::cout << "Number of Customs Agents: " << NUM_AGENTS << "\n" << std::endl;
    std::cout << "Number of Groups per Agent: " << NUM_GROUPS << "\n" << std::endl;
    std::cout << "Random Number Seed: " << SEED << "\n" << std::endl;
    std::cout << "\n" << std::endl;
    std::cout << "-- Simulation Results --\n" << std::endl;
    std::cout << "Total Time Elapsed: " << elapsed << "\n" << std::endl;
    std::cout << "Total Payroll for the day: " << payroll << "\n" << std::endl;
    std::cout << "Average Time Elapsed: " << average << "\n" << std::endl;

    for (int i = 0; i < NUM_AGENTS; i++) {
        delete(d_agents[i]);
        delete(h_agents[i]);
    }
}